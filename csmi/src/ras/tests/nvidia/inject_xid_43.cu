#include "hip/hip_runtime.h"
/*================================================================================

    csmi/src/ras/tests/nvidia/inject_xid_43.cu

  © Copyright IBM Corporation 2015-2017. All Rights Reserved

    This program is licensed under the terms of the Eclipse Public License
    v1.0 as published by the Eclipse Foundation and available at
    http://www.eclipse.org/legal/epl-v10.html

    U.S. Government Users Restricted Rights:  Use, duplication or disclosure
    restricted by GSA ADP Schedule Contract with IBM Corp.

================================================================================*/

#include <stdio.h>

// Compile:
// /usr/local/cuda-8.0/bin/nvcc inject_xid_43.cu -o inject_xid_43

// Run:
// ./inject_xid_43
//
// Run on specific GPU:
// CUDA_VISIBLE_DEVICES=0 ./inject_xid_43
// CUDA_VISIBLE_DEVICES=1 ./inject_xid_43
// CUDA_VISIBLE_DEVICES=2 ./inject_xid_43
// CUDA_VISIBLE_DEVICES=3 ./inject_xid_43

// Sample output:
// dmesg | tail -n 1
// [1991019.552551] NVRM: Xid (PCI:0002:01:00): 43, Ch 00000010, engmask 00000101
// [1991393.936281] NVRM: Xid (PCI:0003:01:00): 43, Ch 00000010, engmask 00000101
// [1991428.078670] NVRM: Xid (PCI:0006:01:00): 43, Ch 00000010, engmask 00000101
// [1991437.093019] NVRM: Xid (PCI:0007:01:00): 43, Ch 00000010, engmask 00000101

__global__ void inject_xid_43(int* gpu_ptr)
{
  int * gpu_ptr2 = NULL;
  gpu_ptr[0] = *gpu_ptr2;
}

int main(void)
{
  int rc(0);
  int* gpu_ptr;
  int size = 8;

  hipMalloc( (void **) &gpu_ptr, size);

  int num_threads = 8;
  int num_blocks = 1;

  inject_xid_43<<<num_blocks,num_threads+1>>>(gpu_ptr);
  
  rc = hipDeviceSynchronize();
  if (rc != hipSuccess)
  {
    fprintf(stderr,"hipDeviceSynchronize() returned status = %d\n",rc);
    return 1;
  }

  return 0;
}

