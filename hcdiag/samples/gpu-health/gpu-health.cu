//#================================================================================
//#
//#    hcdiag/samples/gpu-health.cu
//#
//#  © Copyright IBM Corporation 2015,2016. All Rights Reserved
//#
//#    This program is licensed under the terms of the Eclipse Public License
//#    v1.0 as published by the Eclipse Foundation and available at
//#    http://www.eclipse.org/legal/epl-v10.html
//#
//#    U.S. Government Users Restricted Rights:  Use, duplication or disclosure
//#    restricted by GSA ADP Schedule Contract with IBM Corp.
//#
//#================================================================================
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// 07/26/2018: printing also PCI Bus Id (alda) 

#define MAX_BLOCKS 512
#define THREADS_PER_BLOCK 256

void cuda_dgemm(const char *, const char *, int *, int *, int *, double *, double *, int *, double *, int *, double *, double *, int *);
void cuda_dgemm_free();

#define CUDA_RC(rc) if( (rc) != hipSuccess ) \
  {printf("Error %s at %s line %d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__); exit(1);}

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess )        \
  {printf("Error %s at %s line %d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__-1); exit(1);}

double walltime(void);

__global__ void daxpy(const double alpha, const double * x, double * y, int npts) 
{
   for (int i = blockDim.x * blockIdx.x + threadIdx.x;  i < npts; i += blockDim.x * gridDim.x) y[i] = alpha*x[i] + y[i];
}

int main(int argc, char * argv[])
{
  int i, npts, iter, maxiter, device, numDevices;
  double * __restrict__ x, * __restrict__ y;
  double * dev_x, * dev_y;
  double * Amat, * Bmat, * Cmat;
  int m, n, k, lda, ldb, ldc;
  double alpha, beta, BW;
  double time1, time2;
  hipDeviceProp_t prop;
  char host[240];

  gethostname(host, sizeof(host));

  // set npts large enough to exceed caches for daxpy : 1 GB
  npts = 1024*1024*(1024/8);

  printf("initializing data ...\n\n");

  // set matrix dimensions large enough to reach close to peak Flops
  m = 8192; n = 8192; k = 8192;
  Amat = (double *) malloc(m*k*sizeof(double));
  Bmat = (double *) malloc(k*n*sizeof(double));
  Cmat = (double *) malloc(m*n*sizeof(double));

#pragma omp parallel for
  for (i=0; i<(m*k); i++) Amat[i] = 1.2e-2*((double) (i%100));
#pragma omp parallel for
  for (i=0; i<(k*n); i++) Bmat[i] = 1.5e-3*((double) ((i + 100)%1000));
#pragma omp parallel for
  for (i=0; i<(m*n); i++) Cmat[i] = 1.5e-3*((double) ((i + 500)%1000));

  CUDA_RC(hipGetDeviceCount(&numDevices));
  printf("found %d GPU devices on host %s\n\n", numDevices, host);

  for (device=0; device<numDevices; device++)
  {
     char pciBusId[256];
     CUDA_RC(hipSetDevice(device));
     CUDA_RC(hipGetDeviceProperties(&prop, device));
     CUDA_RC(hipDeviceGetPCIBusId (pciBusId, 256, device));
     printf("checking device %d = %s ... \n", device, prop.name);
     printf("device on PCI Bus ID: %s\n", pciBusId);
     printf("compute capability major = %d, minor = %d\n", prop.major, prop.minor);
 
     // use pinned memory for x, pageable memory for y
     CUDA_RC(hipHostMalloc((void **)&x, npts*sizeof(double), hipHostMallocDefault));
     y = (double *) malloc(npts*sizeof(double));

     CUDA_RC(hipMalloc((void **)&dev_x, npts*sizeof(double)));
     CUDA_RC(hipMalloc((void **)&dev_y, npts*sizeof(double)));

   #pragma omp parallel for
     for (i=0; i<npts; i++) x[i] = (double) (i%10);
   #pragma omp parallel for
     for (i=0; i<npts; i++) y[i] = (double) (i%100);

     alpha = 3.0;
     maxiter = 5;

     time1 = walltime();
     CUDA_RC(hipMemcpy(dev_x, x, npts*sizeof(double), hipMemcpyHostToDevice));
     CUDA_RC(hipDeviceSynchronize());
     time2 = walltime();

     BW = 8.0e-9*((double) npts)/(time2 - time1);
     printf("host to device transfer rate from pinned   memory = %.2lf GB/sec\n", BW);

     time1 = walltime();
     CUDA_RC(hipMemcpy(dev_y, y, npts*sizeof(double), hipMemcpyHostToDevice));
     time2 = walltime();

     BW = 8.0e-9*((double) npts)/(time2 - time1);
     printf("host to device transfer rate from pageable memory = %.2lf GB/sec\n", BW);

     time1 = walltime();
     CUDA_RC(hipMemcpy(x, dev_x, npts*sizeof(double), hipMemcpyDeviceToHost));
     CUDA_RC(hipDeviceSynchronize());
     time2 = walltime();

     BW = 8.0e-9*((double) npts)/(time2 - time1);
     printf("device to host transfer rate from pinned   memory = %.2lf GB/sec\n", BW);

     time1 = walltime();
     CUDA_RC(hipMemcpy(y, dev_y, npts*sizeof(double), hipMemcpyDeviceToHost));
     time2 = walltime();

     BW = 8.0e-9*((double) npts)/(time2 - time1);
     printf("device to host transfer rate from pageable memory = %.2lf GB/sec\n", BW);

     int threadsPerBlock = THREADS_PER_BLOCK;
     int numBlocks = (npts + threadsPerBlock - 1) / threadsPerBlock;
     if (numBlocks > MAX_BLOCKS) numBlocks = MAX_BLOCKS;

     time1 = walltime();
     for (iter=0; iter<maxiter; iter++) {
        daxpy<<<numBlocks, threadsPerBlock>>>(alpha, dev_x, dev_y, npts);
        CUDA_CHECK();
     }
     CUDA_RC(hipDeviceSynchronize());
     time2 = walltime();

     BW = 3.0*8.0e-9*((double) npts)*((double) maxiter)/(time2 - time1);
     printf("GPU daxpy bandwidth = %.2lf GB/sec\n", BW);

     free(y);
     CUDA_RC(hipHostFree(x));
     CUDA_RC(hipFree(dev_x));
     CUDA_RC(hipFree(dev_y));

     beta = 0.0; lda = m; ldb = k; ldc = m;
     cuda_dgemm("N", "N", &m, &n, &k, &alpha, Amat, &lda, Bmat, &ldb, &beta, Cmat, &ldc);
     cuda_dgemm_free();

     printf("\n");
  }

  printf("done\n");

  return 0;
}

double walltime(void)
{
  double elapsed;
  struct timeval tv;
  gettimeofday(&tv,NULL);
  elapsed = ((double) tv.tv_sec) + 1.0e-6*((double) tv.tv_usec);
  return elapsed;
}


// variables for cublas dgemm wrapper
static double * d_A, * d_B, * d_C;
static hipblasHandle_t handle;

// use the Fortran dgemm argument list
void cuda_dgemm(const char * transa, const char * transb, int * m, int * n, int * k, 
                double * alpha, double * A, int * lda, double * B, int * ldb, 
                double * beta, double * C, int * ldc)
{
   int M, N, K, LDA, LDB, LDC;
   int asize, bsize, csize;
   double time1, time2, GFlops;
   hipblasOperation_t opA, opB;

   M = *m; N = *n; K = *k;
   LDA = *lda; LDB = *ldb; LDC = *ldc;

   asize = M*K;
   bsize = K*N;
   csize = M*N;

   hipblasCreate(&handle);
   hipMalloc((void **)&d_A, asize*sizeof(double));
   hipMalloc((void **)&d_B, bsize*sizeof(double));
   hipMalloc((void **)&d_C, csize*sizeof(double));

   hipblasSetVector(asize, sizeof(double), A, 1, d_A, 1);
   hipblasSetVector(bsize, sizeof(double), B, 1, d_B, 1);
   hipblasSetVector(csize, sizeof(double), C, 1, d_C, 1);

   if      (transa[0] == 'n' || transa[0] == 'N') opA = HIPBLAS_OP_N;
   else if (transa[0] == 't' || transa[0] == 'T') opA = HIPBLAS_OP_T;

   if      (transb[0] == 'n' || transb[0] == 'N') opB = HIPBLAS_OP_N;
   else if (transb[0] == 't' || transb[0] == 'T') opB = HIPBLAS_OP_T;

   // call one time outside the timers, then time it
   hipblasDgemm(handle, opA, opB, M, N, K, alpha, d_A, LDA, d_B, LDB, beta, d_C, LDC);
   hipDeviceSynchronize();

   time1 = walltime();
   hipblasDgemm(handle, opA, opB, M, N, K, alpha, d_A, LDA, d_B, LDB, beta, d_C, LDC);
   hipDeviceSynchronize();
   time2 = walltime();
   GFlops = 2.0e-9*((double) M)*((double) N)*((double) K)/(time2 - time1);
   printf("GPU dgemm TFlops = %.3lf\n", 1.0e-3*GFlops);

   hipMemcpy(C, d_C, csize*sizeof(double), hipMemcpyDeviceToHost);

   return;
}

void cuda_dgemm_free()
{
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   hipblasDestroy(handle);
   return;
}
