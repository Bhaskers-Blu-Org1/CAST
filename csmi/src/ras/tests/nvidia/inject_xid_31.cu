#include "hip/hip_runtime.h"
/*================================================================================

    csmi/src/ras/tests/nvidia/inject_xid_31.cu

  © Copyright IBM Corporation 2015-2017. All Rights Reserved

    This program is licensed under the terms of the Eclipse Public License
    v1.0 as published by the Eclipse Foundation and available at
    http://www.eclipse.org/legal/epl-v10.html

    U.S. Government Users Restricted Rights:  Use, duplication or disclosure
    restricted by GSA ADP Schedule Contract with IBM Corp.

================================================================================*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
 
// Compile:
// /usr/local/cuda-8.0/bin/nvcc inject_xid_31.cu -o inject_xid_31 

// Run:
// ./inject_xid_31
// Run on specific GPU:
// CUDA_VISIBLE_DEVICES=0 ./inject_xid_31
// CUDA_VISIBLE_DEVICES=1 ./inject_xid_31
// CUDA_VISIBLE_DEVICES=2 ./inject_xid_31
// CUDA_VISIBLE_DEVICES=3 ./inject_xid_31

// Sample output:
// dmesg | tail -n 1
// [1992033.683091] NVRM: Xid (PCI:0002:01:00): 31, Ch 00000010, engmask 00000101, intr 10000000
// [1992005.493582] NVRM: Xid (PCI:0003:01:00): 31, Ch 00000010, engmask 00000101, intr 10000000
// [1992013.187501] NVRM: Xid (PCI:0006:01:00): 31, Ch 00000010, engmask 00000101, intr 10000000
// [1992019.406216] NVRM: Xid (PCI:0007:01:00): 31, Ch 00000010, engmask 00000101, intr 10000000

__global__ void
genXid0()
{
  double *p0 = NULL;
 
  int ii = blockIdx.x * blockDim.x + threadIdx.x;
  p0[ii] = 0.0;
}
 
#define VECSIZE 1000
struct hipDeviceProp_t cudaDeviceProps;
 
int main(int argc, char** argv)
{
  int device_count(0);
  int rc(0);
 
  hipGetDeviceCount(&device_count);
  printf("device count = %d\n",device_count);
 
  for (int i = 0; i < device_count; i++)
  {
    hipGetDeviceProperties(&cudaDeviceProps,i);
    printf("device name = %s\n",cudaDeviceProps.name);
  } 

  dim3 grid(64,64);
  dim3 block(16,16);
  genXid0<<<grid,block>>>();

  rc = hipDeviceSynchronize();
  if (rc != hipSuccess)
  {
    fprintf(stderr,"hipDeviceSynchronize() returned status = %d\n",rc);
    return 1;
  }
 
  return 0;
}
